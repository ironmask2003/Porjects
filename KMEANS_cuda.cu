#include "hip/hip_runtime.h"
/*
 * k-Means clustering algorithm
 *
 * CUDA version
 *
 * Parallel computing (Degree in Computer Engineering)
 * 2022/2023
 *
 * Version: 1.0
 *
 * (c) 2022 Diego García-Álvarez, Arturo Gonzalez-Escribano
 * Grupo Trasgo, Universidad de Valladolid (Spain)
 *
 * This work is licensed under a Creative Commons Attribution-ShareAlike 4.0 International License.
 * https://creativecommons.org/licenses/by-sa/4.0/
 */
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <float.h>
#include <omp.h>
#include <hip/hip_runtime.h>


#define MAXLINE 2000
#define MAXCAD 200

//Macros
#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))

/*
 * Macros to show errors when calling a CUDA library function,
 * or after launching a kernel
 */
#define CHECK_CUDA_CALL( a )	{ \
	hipError_t ok = a; \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}
#define CHECK_CUDA_LAST()	{ \
	hipError_t ok = hipGetLastError(); \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}

/* 
Function showFileError: It displays the corresponding error during file reading.
*/
void showFileError(int error, char* filename)
{
	printf("Error\n");
	switch (error)
	{
		case -1:
			fprintf(stderr,"\tFile %s has too many columns.\n", filename);
			fprintf(stderr,"\tThe maximum number of columns has been exceeded. MAXLINE: %d.\n", MAXLINE);
			break;
		case -2:
			fprintf(stderr,"Error reading file: %s.\n", filename);
			break;
		case -3:
			fprintf(stderr,"Error writing file: %s.\n", filename);
			break;
	}
	fflush(stderr);	
}

/* 
Function readInput: It reads the file to determine the number of rows and columns.
*/
int readInput(char* filename, int *lines, int *samples)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int contlines, contsamples = 0;
    
    contlines = 0;

    if ((fp=fopen(filename,"r"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL) 
		{
			if (strchr(line, '\n') == NULL)
			{
				return -1;
			}
            contlines++;       
            ptr = strtok(line, delim);
            contsamples = 0;
            while(ptr != NULL)
            {
            	contsamples++;
				ptr = strtok(NULL, delim);
	    	}	    
        }
        fclose(fp);
        *lines = contlines;
        *samples = contsamples;  
        return 0;
    }
    else
	{
    	return -2;
	}
}

/* 
Function readInput2: It loads data from file.
*/
int readInput2(char* filename, float* data)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int i = 0;
    
    if ((fp=fopen(filename,"rt"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL)
        {         
            ptr = strtok(line, delim);
            while(ptr != NULL)
            {
            	data[i] = atof(ptr);
            	i++;
				ptr = strtok(NULL, delim);
	   		}
	    }
        fclose(fp);
        return 0;
    }
    else
	{
    	return -2; //No file found
	}
}

/* 
Function writeResult: It writes in the output file the cluster of each sample (point).
*/
int writeResult(int *classMap, int lines, const char* filename)
{	
    FILE *fp;
    
    if ((fp=fopen(filename,"wt"))!=NULL)
    {
        for(int i=0; i<lines; i++)
        {
        	fprintf(fp,"%d\n",classMap[i]);
        }
        fclose(fp);  
   
        return 0;
    }
    else
	{
    	return -3; //No file found
	}
}

/*

Function initCentroids: This function copies the values of the initial centroids, using their 
position in the input data structure as a reference map.
*/
void initCentroids(const float *data, float* centroids, int* centroidPos, int samples, int K)
{
	int i;
	int idx;
	for(i=0; i<K; i++)
	{
		idx = centroidPos[i];
		memcpy(&centroids[i*samples], &data[idx*samples], (samples*sizeof(float)));
	}
}

/*
Function euclideanDistance: Euclidean distance
This function could be modified
*/
__device__ float euclideanDistance(float *point, float *center, int samples)
{
	float dist=0.0;
	for(int i=0; i<samples; i++) 
	{
		dist+= (point[i]-center[i])*(point[i]-center[i]);
	}
	dist = sqrt(dist);
	return(dist);
}

/*
Function zeroFloatMatriz: Set matrix elements to 0
This function could be modified
*/
void zeroFloatMatriz(float *matrix, int rows, int columns)
{
	int i,j;
	for (i=0; i<rows; i++)
		for (j=0; j<columns; j++)
			matrix[i*columns+j] = 0.0;	
}

/*
Function zeroIntArray: Set array elements to 0
This function could be modified
*/
void zeroIntArray(int *array, int size)
{
	int i;
	for (i=0; i<size; i++)
		array[i] = 0;	
}


// Variabili global
__constant__ int gpu_samples;
__constant__ int gpu_K;
__constant__ int gpu_lines;

// Funzione che assegna ad ogni punto il centroide più vicino
__global__ void assign_centroids(float *d_data, float *d_centroids, int *d_classMap, int* changes, int* class_var)
{
	int thread_index = (blockIdx.y * gridDim.x * blockDim.x * blockDim.y) + (blockIdx.x * blockDim.x * blockDim.y) +
							(threadIdx.y * blockDim.x) +
							threadIdx.x;

	if(thread_index < gpu_lines)
	{
		class_var=1;
		float dist, minDist=FLT_MAX;

		for(int j=0; j<gpu_K; j++)
		{
			dist=euclideanDistance(&d_data[thread_index*gpu_samples], &d_centroids[j*gpu_samples], gpu_samples);

			if(dist < minDist)
			{
				minDist=dist;
				class_var=j+1;
			}
		}

		if(d_classMap[thread_index]!=class_var)
		{
			atomicAdd(changes, 1);
		}

		d_classMap[thread_index]=class_var;
	}
}

int main(int argc, char* argv[])
{

	//START CLOCK***************************************
	double start, end;
	start = omp_get_wtime();
	//**************************************************
	/*
	* PARAMETERS
	*
	* argv[1]: Input data file
	* argv[2]: Number of clusters
	* argv[3]: Maximum number of iterations of the method. Algorithm termination condition.
	* argv[4]: Minimum percentage of class changes. Algorithm termination condition.
	*          If between one iteration and the next, the percentage of class changes is less than
	*          this percentage, the algorithm stops.
	* argv[5]: Precision in the centroid distance after the update.
	*          It is an algorithm termination condition. If between one iteration of the algorithm 
	*          and the next, the maximum distance between centroids is less than this precision, the
	*          algorithm stops.
	* argv[6]: Output file. Class assigned to each point of the input file.
	* */
	if(argc !=  7)
	{
		fprintf(stderr,"EXECUTION ERROR K-MEANS: Parameters are not correct.\n");
		fprintf(stderr,"./KMEANS [Input Filename] [Number of clusters] [Number of iterations] [Number of changes] [Threshold] [Output data file]\n");
		fflush(stderr);
		exit(-1);
	}

	// Reading the input data
	// lines = number of points; samples = number of dimensions per point
	int lines = 0, samples= 0;  
	
	int error = readInput(argv[1], &lines, &samples);
	if(error != 0)
	{
		showFileError(error,argv[1]);
		exit(error);
	}
	
	float *data = (float*)calloc(lines*samples,sizeof(float));
	if (data == NULL)
	{
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}
	error = readInput2(argv[1], data);
	if(error != 0)
	{
		showFileError(error,argv[1]);
		exit(error);
	}

	// Parameters
	int K=atoi(argv[2]); 
	int maxIterations=atoi(argv[3]);
	int minChanges= (int)(lines*atof(argv[4])/100.0);
	float maxThreshold=atof(argv[5]);

	int *centroidPos = (int*)calloc(K,sizeof(int));
	float *centroids = (float*)calloc(K*samples,sizeof(float));
	int *classMap = (int*)calloc(lines,sizeof(int));

    if (centroidPos == NULL || centroids == NULL || classMap == NULL)
	{
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}

	// Initial centrodis
	srand(0);
	int i;
	for(i=0; i<K; i++) 
		centroidPos[i]=rand()%lines;
	
	// Loading the array of initial centroids with the data from the array data
	// The centroids are points stored in the data array.
	initCentroids(data, centroids, centroidPos, samples, K);


	printf("\n\tData file: %s \n\tPoints: %d\n\tDimensions: %d\n", argv[1], lines, samples);
	printf("\tNumber of clusters: %d\n", K);
	printf("\tMaximum number of iterations: %d\n", maxIterations);
	printf("\tMinimum number of changes: %d [%g%% of %d points]\n", minChanges, atof(argv[4]), lines);
	printf("\tMaximum centroid precision: %f\n", maxThreshold);
	
	//END CLOCK*****************************************
	end = omp_get_wtime();
	printf("\nMemory allocation: %f seconds\n", end - start);
	fflush(stdout);

	CHECK_CUDA_CALL( hipSetDevice(0) );
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
	//**************************************************
	//START CLOCK***************************************
	start = omp_get_wtime();
	//**************************************************
	char *outputMsg = (char *)calloc(10000,sizeof(char));
	char line[100];

	int j;
	int class_var;
	float dist, minDist;
	int it=0;
	int changes = 0;
	float maxDist;

	//pointPerClass: number of points classified in each class
	//auxCentroids: mean of the points in each class
	int *pointsPerClass = (int *)malloc(K*sizeof(int));
	float *auxCentroids = (float*)malloc(K*samples*sizeof(float));
	float *distCentroids = (float*)malloc(K*sizeof(float)); 
	if (pointsPerClass == NULL || auxCentroids == NULL || distCentroids == NULL)
	{
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 */

	// Inizializzazione delle costanti per le gpu
	CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gpu_K), &K, sizeof(int)));
	CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gpu_samples), &samples, sizeof(int)));
	CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gpu_lines), &lines, sizeof(int)));

	// Adapt to the number of points
	int pts_grid_size = samples / (32 * 32) + 1;
	int K_grid_size = K / (32 * 32) + 1;

	// Set carveout to be of maximum size available
	int carveout = cudaSharedmemCarveoutMaxShared;

	CHECK_CUDA_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(assign_centroids), hipFuncAttributePreferredSharedMemoryCarveout, carveout));

	dim3 gen_block(32, 32);
	dim3 dyn_grid_pts(pts_grid_size);
	dim3 dyn_grid_cent(K_grid_size);

	// Copy data to device
	float *d_data;
	int *d_classMap;
	float *d_centroids;
	int *d_pointsPerClass;
	float *d_auxCentroids;
	float *d_distCentroids;

	CHECK_CUDA_CALL( hipMalloc(&d_data, lines*samples*sizeof(float)) );
	CHECK_CUDA_CALL( hipMemcpy(d_data, data, lines*samples*sizeof(float), hipMemcpyHostToDevice) );

	CHECK_CUDA_CALL( hipMalloc(&d_classMap, lines*sizeof(int)) );
	CHECK_CUDA_CALL( hipMemcpy(d_classMap, classMap, lines*sizeof(int), hipMemcpyHostToDevice) );
	
	CHECK_CUDA_CALL( hipMalloc(&d_centroids, K*samples*sizeof(float)) );
	CHECK_CUDA_CALL( hipMemcpy(d_centroids, centroids, K*samples*sizeof(float), hipMemcpyHostToDevice) );
	
	CHECK_CUDA_CALL( hipMalloc(&d_pointsPerClass, K*sizeof(int)) );
	CHECK_CUDA_CALL( hipMemcpy(d_pointsPerClass, pointsPerClass, K*sizeof(int), hipMemcpyHostToDevice) );
	
	CHECK_CUDA_CALL( hipMalloc(&d_auxCentroids, K*samples*sizeof(float)) );
	CHECK_CUDA_CALL( hipMemcpy(d_auxCentroids, auxCentroids, K*samples*sizeof(float), hipMemcpyHostToDevice) );
	
	CHECK_CUDA_CALL( hipMalloc(&d_distCentroids, K*sizeof(float)) );
	CHECK_CUDA_CALL( hipMemcpy(d_distCentroids, distCentroids, K*sizeof(float), hipMemcpyHostToDevice) );

	do{
		it++;
	
		//1. Calculate the distance from each point to the centroid
		//Assign each point to the nearest centroid.

		CHECK_CUDA_CALL(hipMemset(changes, 0, sizeof(int)));

		// Synschronize
		CHECK_CUDA_CALL(hipDeviceSynchronize());

		assign_centroids<<<dyn_grid_pts, gen_block, K * lines * sizeof(float)>>>(d_data, d_centroids, d_classMap, changes, class_var);

		// Syncronize
		CHECK_CUDA_CALL(hipDeviceSynchronize());

		CHECK_CUDA_CALL( hipMemset(d_pointsPerClass, 0, K*sizeof(int)) );
		CHECK_CUDA_CALL( hipMemset(d_auxCentroids, 0, K*samples*sizeof(float)) );

		for(i=0; i<lines; i++) 
		{
			class_var=classMap[i];
			pointsPerClass[class_var-1] = pointsPerClass[class_var-1] +1;
			for(j=0; j<samples; j++){
				auxCentroids[(class_var-1)*samples+j] += data[i*samples+j];
			}
		}

		for(i=0; i<K; i++) 
		{
			for(j=0; j<samples; j++){
				auxCentroids[i*samples+j] /= pointsPerClass[i];
			}
		}
		
		maxDist=FLT_MIN;
		for(i=0; i<K; i++){
			distCentroids[i]=euclideanDistance(&centroids[i*samples], &auxCentroids[i*samples], samples);
			if(distCentroids[i]>maxDist) {
				maxDist=distCentroids[i];
			}
		}
		memcpy(centroids, auxCentroids, (K*samples*sizeof(float)));
		
		sprintf(line,"\n[%d] Cluster changes: %d\tMax. centroid distance: %f", it, changes, maxDist);
		outputMsg = strcat(outputMsg,line);

	} while((changes>minChanges) && (it<maxIterations) && (maxDist>maxThreshold));

/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */
	// Output and termination conditions
	printf("%s",outputMsg);	

	CHECK_CUDA_CALL( hipDeviceSynchronize() );

	//END CLOCK*****************************************
	end = omp_get_wtime();
	printf("\nComputation: %f seconds", end - start);
	fflush(stdout);
	//**************************************************
	//START CLOCK***************************************
	start = omp_get_wtime();
	//**************************************************

	

	if (changes <= minChanges) {
		printf("\n\nTermination condition:\nMinimum number of changes reached: %d [%d]", changes, minChanges);
	}
	else if (it >= maxIterations) {
		printf("\n\nTermination condition:\nMaximum number of iterations reached: %d [%d]", it, maxIterations);
	}
	else {
		printf("\n\nTermination condition:\nCentroid update precision reached: %g [%g]", maxDist, maxThreshold);
	}	

	// Writing the classification of each point to the output file.
	error = writeResult(classMap, lines, argv[6]);
	if(error != 0)
	{
		showFileError(error, argv[6]);
		exit(error);
	}

	//Free memory
	free(data);
	free(classMap);
	free(centroidPos);
	free(centroids);
	free(distCentroids);
	free(pointsPerClass);
	free(auxCentroids);

	//END CLOCK*****************************************
	end = omp_get_wtime();
	printf("\n\nMemory deallocation: %f seconds\n", end - start);
	fflush(stdout);
	//***************************************************/
	return 0;
}
