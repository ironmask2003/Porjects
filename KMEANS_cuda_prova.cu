/*
 * k-Means clustering algorithm
 *
 * CUDA version
 *
 * Parallel computing (Degree in Computer Engineering)
 * 2022/2023
 *
 * Version: 1.0
 *
 * (c) 2022 Diego García-Álvarez, Arturo Gonzalez-Escribano
 * Grupo Trasgo, Universidad de Valladolid (Spain)
 *
 * This work is licensed under a Creative Commons Attribution-ShareAlike 4.0 International License.
 * https://creativecommons.org/licenses/by-sa/4.0/
 */
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <float.h>
#include <omp.h>
#include <hip/hip_runtime.h>


#define MAXLINE 2000
#define MAXCAD 200

//Macros
#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))

/*
 * Macros to show errors when calling a CUDA library function,
 * or after launching a kernel
 */
#define CHECK_CUDA_CALL( a )	{ \
	hipError_t ok = a; \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}
#define CHECK_CUDA_LAST()	{ \
	hipError_t ok = hipGetLastError(); \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}

/* 
Function showFileError: It displays the corresponding error during file reading.
*/
void showFileError(int error, char* filename)
{
	printf("Error\n");
	switch (error)
	{
		case -1:
			fprintf(stderr,"\tFile %s has too many columns.\n", filename);
			fprintf(stderr,"\tThe maximum number of columns has been exceeded. MAXLINE: %d.\n", MAXLINE);
			break;
		case -2:
			fprintf(stderr,"Error reading file: %s.\n", filename);
			break;
		case -3:
			fprintf(stderr,"Error writing file: %s.\n", filename);
			break;
	}
	fflush(stderr);	
}

/* 
Function readInput: It reads the file to determine the number of rows and columns.
*/
int readInput(char* filename, int *lines, int *samples)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int contlines, contsamples = 0;
    
    contlines = 0;

    if ((fp=fopen(filename,"r"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL) 
		{
			if (strchr(line, '\n') == NULL)
			{
				return -1;
			}
            contlines++;       
            ptr = strtok(line, delim);
            contsamples = 0;
            while(ptr != NULL)
            {
            	contsamples++;
				ptr = strtok(NULL, delim);
	    	}	    
        }
        fclose(fp);
        *lines = contlines;
        *samples = contsamples;  
        return 0;
    }
    else
	{
    	return -2;
	}
}

/* 
Function readInput2: It loads data from file.
*/
int readInput2(char* filename, float* data)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int i = 0;
    
    if ((fp=fopen(filename,"rt"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL)
        {         
            ptr = strtok(line, delim);
            while(ptr != NULL)
            {
            	data[i] = atof(ptr);
            	i++;
				ptr = strtok(NULL, delim);
	   		}
	    }
        fclose(fp);
        return 0;
    }
    else
	{
    	return -2; //No file found
	}
}

/* 
Function writeResult: It writes in the output file the cluster of each sample (point).
*/
int writeResult(int *classMap, int lines, const char* filename)
{	
    FILE *fp;
    
    if ((fp=fopen(filename,"wt"))!=NULL)
    {
        for(int i=0; i<lines; i++)
        {
        	fprintf(fp,"%d\n",classMap[i]);
        }
        fclose(fp);  
   
        return 0;
    }
    else
	{
    	return -3; //No file found
	}
}

/*

Function initCentroids: This function copies the values of the initial centroids, using their 
position in the input data structure as a reference map.
*/
void initCentroids(const float *data, float* centroids, int* centroidPos, int samples, int K)
{
	int i;
	int idx;
	for(i=0; i<K; i++)
	{
		idx = centroidPos[i];
		memcpy(&centroids[i*samples], &data[idx*samples], (samples*sizeof(float)));
	}
}

/*
Function euclideanDistance: Euclidean distance
This function could be modified
*/
__device__ float euclideanDistance_gpu(float *point, float *center, int samples)
{
	float dist=0.0;
	for(int i=0; i<samples; i++) 
	{
		dist+= (point[i]-center[i])*(point[i]-center[i]);
	}
	dist = sqrt(dist);
	return(dist);
}

float euclideanDistance(float *point, float *center, int samples)
{
	float dist=0.0;
	for(int i=0; i<samples; i++) 
	{
		dist+= (point[i]-center[i])*(point[i]-center[i]);
	}
	dist = sqrt(dist);
	return(dist);
}

/*
Function zeroFloatMatriz: Set matrix elements to 0
This function could be modified
*/
void zeroFloatMatriz(float *matrix, int rows, int columns)
{
	int i,j;
	for (i=0; i<rows; i++)
		for (j=0; j<columns; j++)
			matrix[i*columns+j] = 0.0;	
}

/*
Function zeroIntArray: Set array elements to 0
This function could be modified
*/
void zeroIntArray(int *array, int size)
{
	int i;
	for (i=0; i<size; i++)
		array[i] = 0;	
}

// Variabili globali
__constant__ int d_K;
__constant__ int d_samples;
__constant__ int d_lines;

__global__ void assign_centroids(float *d_data, float *d_centroids, int *d_classMap, int* changes){
    int thread_index = (blockIdx.y * gridDim.x * blockDim.x * blockDim.y) + (blockIdx.x * blockDim.x * blockDim.y) +
							(threadIdx.y * blockDim.x) +
							threadIdx.x;

	if(thread_index < d_lines) {
		int class_var = 1;
		float dist, minDist=FLT_MAX;

		for(int j=0; j<d_K; j++)
		{
			dist=euclideanDistance_gpu(&d_data[thread_index*d_samples], &d_centroids[j*d_samples], d_samples);

			if(dist < minDist)
			{
				minDist=dist;
				class_var=j+1;
			}
		}

		if(d_classMap[thread_index]!=class_var)
		{
			atomicAdd(changes, 1);
		}

		d_classMap[thread_index]=class_var;
    }
}

int main(int argc, char* argv[])
{

	//START CLOCK***************************************
	double start, end;
	start = omp_get_wtime();
	//**************************************************
	/*
	* PARAMETERS
	*
	* argv[1]: Input data file
	* argv[2]: Number of clusters
	* argv[3]: Maximum number of iterations of the method. Algorithm termination condition.
	* argv[4]: Minimum percentage of class changes. Algorithm termination condition.
	*          If between one iteration and the next, the percentage of class changes is less than
	*          this percentage, the algorithm stops.
	* argv[5]: Precision in the centroid distance after the update.
	*          It is an algorithm termination condition. If between one iteration of the algorithm 
	*          and the next, the maximum distance between centroids is less than this precision, the
	*          algorithm stops.
	* argv[6]: Output file. Class assigned to each point of the input file.
	* */
	if(argc !=  7)
	{
		fprintf(stderr,"EXECUTION ERROR K-MEANS: Parameters are not correct.\n");
		fprintf(stderr,"./KMEANS [Input Filename] [Number of clusters] [Number of iterations] [Number of changes] [Threshold] [Output data file]\n");
		fflush(stderr);
		exit(-1);
	}

	// Reading the input data
	// lines = number of points; samples = number of dimensions per point
	int lines = 0, samples= 0;  
	
	int error = readInput(argv[1], &lines, &samples);
	if(error != 0)
	{
		showFileError(error,argv[1]);
		exit(error);
	}
	
	float *data = (float*)calloc(lines*samples,sizeof(float));
	if (data == NULL)
	{
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}
	error = readInput2(argv[1], data);
	if(error != 0)
	{
		showFileError(error,argv[1]);
		exit(error);
	}

	// Parameters
	int K=atoi(argv[2]); 
	int maxIterations=atoi(argv[3]);
	int minChanges= (int)(lines*atof(argv[4])/100.0);
	float maxThreshold=atof(argv[5]);

	int *centroidPos = (int*)calloc(K,sizeof(int));
	float *centroids = (float*)calloc(K*samples,sizeof(float));
	int *classMap = (int*)calloc(lines,sizeof(int));

    if (centroidPos == NULL || centroids == NULL || classMap == NULL)
	{
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}

	// Initial centrodis
	srand(0);
	int i;
	for(i=0; i<K; i++) 
		centroidPos[i]=rand()%lines;
	
	// Loading the array of initial centroids with the data from the array data
	// The centroids are points stored in the data array.
	initCentroids(data, centroids, centroidPos, samples, K);


	printf("\n\tData file: %s \n\tPoints: %d\n\tDimensions: %d\n", argv[1], lines, samples);
	printf("\tNumber of clusters: %d\n", K);
	printf("\tMaximum number of iterations: %d\n", maxIterations);
	printf("\tMinimum number of changes: %d [%g%% of %d points]\n", minChanges, atof(argv[4]), lines);
	printf("\tMaximum centroid precision: %f\n", maxThreshold);
	
	//END CLOCK*****************************************
	end = omp_get_wtime();
	printf("\nMemory allocation: %f seconds\n", end - start);
	fflush(stdout);

	CHECK_CUDA_CALL( hipSetDevice(0) );
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
	//**************************************************
	//START CLOCK***************************************
	start = omp_get_wtime();
	//**************************************************
	char *outputMsg = (char *)calloc(10000,sizeof(char));
	char line[100];

	int j;
	int class_var;
	float dist, minDist;
	int it=0;
	int changes = 0;
	float maxDist;

	//pointPerClass: number of points classified in each class
	//auxCentroids: mean of the points in each class
	int *pointsPerClass = (int *)malloc(K*sizeof(int));
	float *auxCentroids = (float*)malloc(K*samples*sizeof(float));
	float *distCentroids = (float*)malloc(K*sizeof(float)); 
	if (pointsPerClass == NULL || auxCentroids == NULL || distCentroids == NULL)
	{
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 */

    int pts_grid_size = samples / (32 * 32) + 1;
	int K_grid_size = K / (32 * 32) + 1;

	dim3 gen_block(32, 32);
	dim3 dyn_grid_pts(pts_grid_size);
	dim3 dyn_grid_cent(K_grid_size);

    // Inizializzazione delle costanti per le gpu
    CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_K), &K, sizeof(int)));
    CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_samples), &samples, sizeof(int)));
    CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_lines), &lines, sizeof(int)));

    // Definizione delle variabili da passare al kernel
    float* d_data;
    float* d_centroids;
    int* d_classMap;
    int* d_changes;

    // Allocazione della memoria sul device
    CHECK_CUDA_CALL( hipMalloc((void**)&d_data, lines*samples*sizeof(float)) );
    CHECK_CUDA_CALL( hipMemcpy(d_data, data, lines*samples*sizeof(float), hipMemcpyHostToDevice) );

    CHECK_CUDA_CALL( hipMalloc((void**)&d_centroids, K*samples*sizeof(float)) );
    CHECK_CUDA_CALL( hipMemcpy(d_centroids, centroids, K*samples*sizeof(float), hipMemcpyHostToDevice) );

    CHECK_CUDA_CALL( hipMalloc((void**)&d_classMap, lines*sizeof(int)) );
    CHECK_CUDA_CALL( hipMemcpy(d_classMap, classMap, lines*sizeof(int), hipMemcpyHostToDevice) );

    CHECK_CUDA_CALL( hipMalloc((void**)&d_changes, sizeof(int)) );

	do{
		it++;
	
		//1. Calculate the distance from each point to the centroid
		//Assign each point to the nearest centroid.
		CHECK_CUDA_CALL( hipMemset(d_changes, 0, sizeof(int)) );

        CHECK_CUDA_CALL( hipMemcpy(d_centroids, centroids, K*samples*sizeof(float), hipMemcpyHostToDevice) );
		CHECK_CUDA_CALL( hipMemcpy(d_classMap, classMap, lines*sizeof(int), hipMemcpyHostToDevice) );

        // Synschronize
		CHECK_CUDA_CALL(hipDeviceSynchronize());

        assign_centroids<<<dyn_grid_pts, gen_block>>>(d_data, d_centroids, d_classMap, d_changes);
		CHECK_CUDA_LAST();

        // Syncronize
		CHECK_CUDA_CALL(hipDeviceSynchronize());

        CHECK_CUDA_CALL( hipMemcpy(&changes, d_changes, sizeof(int), hipMemcpyDeviceToHost) );
        CHECK_CUDA_CALL( hipMemcpy(classMap, d_classMap, lines*sizeof(int), hipMemcpyDeviceToHost) );

		// 2. Recalculates the centroids: calculates the mean within each cluster
		zeroIntArray(pointsPerClass,K);
		zeroFloatMatriz(auxCentroids,K,samples);

		for(i=0; i<lines; i++) 
		{
			class_var=classMap[i];
			pointsPerClass[class_var-1] = pointsPerClass[class_var-1] +1;
			for(j=0; j<samples; j++){
				auxCentroids[(class_var-1)*samples+j] += data[i*samples+j];
			}
		}

		for(i=0; i<K; i++) 
		{
			for(j=0; j<samples; j++){
				auxCentroids[i*samples+j] /= pointsPerClass[i];
			}
		}
		
		maxDist=FLT_MIN;
		for(i=0; i<K; i++){
			distCentroids[i]=euclideanDistance(&centroids[i*samples], &auxCentroids[i*samples], samples);
			if(distCentroids[i]>maxDist) {
				maxDist=distCentroids[i];
			}
		}
		memcpy(centroids, auxCentroids, (K*samples*sizeof(float)));
		
		sprintf(line,"\n[%d] Cluster changes: %d\tMax. centroid distance: %f", it, changes, maxDist);
		outputMsg = strcat(outputMsg,line);

        // Syncronize
		CHECK_CUDA_CALL(hipDeviceSynchronize());

	} while((changes>minChanges) && (it<maxIterations) && (maxDist>maxThreshold));
	
	CHECK_CUDA_CALL(hipFree(d_data));
	CHECK_CUDA_CALL(hipFree(d_centroids));
	CHECK_CUDA_CALL(hipFree(d_classMap));
	CHECK_CUDA_CALL(hipFree(d_changes));
/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */
	// Output and termination conditions
	printf("%s",outputMsg);	

	CHECK_CUDA_CALL( hipDeviceSynchronize() );

	//END CLOCK*****************************************
	end = omp_get_wtime();
	printf("\nComputation: %f seconds", end - start);
	fflush(stdout);
	printf("\nciao");
	//**************************************************
	//START CLOCK***************************************
	start = omp_get_wtime();
	//**************************************************

	

	if (changes <= minChanges) {
		printf("\n\nTermination condition:\nMinimum number of changes reached: %d [%d]", changes, minChanges);
	}
	else if (it >= maxIterations) {
		printf("\n\nTermination condition:\nMaximum number of iterations reached: %d [%d]", it, maxIterations);
	}
	else {
		printf("\n\nTermination condition:\nCentroid update precision reached: %g [%g]", maxDist, maxThreshold);
	}	

	// Writing the classification of each point to the output file.
	error = writeResult(classMap, lines, argv[6]);
	if(error != 0)
	{
		showFileError(error, argv[6]);
		exit(error);
	}

	//Free memory
	free(data);
	free(classMap);
	free(centroidPos);
	free(centroids);
	free(distCentroids);
	free(pointsPerClass);
	free(auxCentroids);

	//END CLOCK*****************************************
	end = omp_get_wtime();
	printf("\n\nMemory deallocation: %f seconds\n", end - start);
	fflush(stdout);
	//***************************************************/
	return 0;
}
